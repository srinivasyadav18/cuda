#include <stdio.h>
#include <string>
#include <vector>
#include <algorithm>
#include <numeric>
#include <iostream>

#include <hip/hip_runtime.h>


__global__ void left_shift_kernel(int *a, const int N){
    int idx = threadIdx.x;
    if (idx < N - 1){
        int temp = a[idx + 1];
        __syncthreads();
        a[idx] = temp;
        __syncthreads();
    }
}

int main(){
    int N = 1 << 10;
    size_t size = N * sizeof(int);

    int *a = new int[N];
    std::iota(a, a + N, 0);

    for (int i = 0; i < N; i++){
        printf("%d ", a[i]);
    }
    printf("\n");

    int *dev_a;
    hipMalloc(&dev_a, size);
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

    left_shift_kernel<<<4, 256>>> (dev_a, N);

    hipMemcpy(a, dev_a, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++){
        printf("%d ", a[i]);
    }
    printf("\n");

    hipFree(dev_a);
    delete[] a;
}