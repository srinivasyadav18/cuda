#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <vector>
#include <algorithm>
#include <numeric>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void mat_mul_kernel(const int *a, const int *b, int *c, const int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for (int k = 0; k < N; k++) {
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

int main(){
    // 512 * 512
    int N = 1 << 9;
    int size = N * N * sizeof(int);

    // Allocate memory on Host 
    int *a = new int[N * N];
    int *b = new int[N * N];
    int *c = new int[N * N];

    // Initialising arrays
    std::iota(a, a + N*N, 0);
    std::iota(b, b + N*N, 1);

    int *dev_a, *dev_b, *dev_c;
    // Allocating memory on device
    hipMalloc(dev_a, size);
    hipMalloc(dev_b, size);
    hipMalloc(dev_c, size);

    // Copy data from host to device
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, size, hipMemcpyHostToDevice);

    // lauching a kernel of 512 blocks each containing 512 threads
    mat_mul_kernel<<<512, 512>>> (dev_a, dev_b, dev_c, N);

    // copy result from device to Host
    hipMemcpy(a, dev_a, size, hipMemcpyDeviceToHost);
    hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    // Free the allocated memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    delete[] a;
    delete[] b;
    delete[] c;
}