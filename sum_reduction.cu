#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <vector>
#include <algorithm>
#include <numeric>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define SHARDED_MEM_SIZE 256

__global__ void sum_reduce_kernel(int *a, int * res, const int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int parital_sum[SHARDED_MEM_SIZE];
    parital_sum[threadIdx.x] = a[idx];
    
    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2){
        if (threadIdx.x % (2 * stride == 0)){
            parital_sum[threadIdx.x] += parital_sum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0){
        res[blockIdx.x] = parital_sum[0];
    }

}

int main(){
    const int N = 1 << 16;
    size_t size = N * sizeof(int);

    int *a = new int[N];
    int *res = new int[N];
    memset(a, 1, N);
    memset(a, 0, N);

    int *dev_a, *dev_res;
    hipMalloc(&dev_a, size);
    hipMalloc(&dev_res, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 256;
    const int GRID_SIZE = N / BLOCK_SIZE;

    sum_reduce_kernel<<<GRID_SIZE, BLOCK_SIZE>>> (dev_a, dev_res);
    sum_reduce_kernel<<<GRID_SIZE, BLOCK_SIZE>>> (dev_res, dev_res);

    hipMemcpy(res, dev_res, size, hipMemcpyDeviceToHost);

    std::cout << "Sum = " << res[0]  << std::endl;
    return 0;
}