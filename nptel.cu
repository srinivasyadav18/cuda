#include <stdio.h>
#include <hip/hip_runtime.h>


#define LOG_INPUT if(0)
#define LOG_OUTPUT if(1)
#define LOG if(0)


__global__ void hadamard(float *A, float *B, float *C, int M, int N)
{
    // Complete the kernel code snippet
	int i=threadIdx.x+blockDim.x*blockIdx.x;
		if (i<M*N){
		C[i]=B[i]*A[i];
		}
}

/**
 * Host main routine
 */
void print_matrix(float *A,int m,int n)
{
    for(int i =0;i<m;i++)
    {
        for(int j=0;j<n;j++)
            printf("%.2f ",A[i*n+j]);
        printf("\n");
    }

}
int main(void)
{
  
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    
    //:wq
    int t; //number of test cases
    scanf("%d",&t);
    while(t--)
    {
        int m,n;
        scanf("%d %d",&m,&n);
        size_t size = m*n * sizeof(float);
        LOG printf("[Hadamard product of two matrices ]\n");
//	printf("scanning ...");
        // Allocate the host input vector A
        
        // Allocate the host input vector B
        
        // Allocate the host output vector C
        

        // Verify that allocations succeeded
       	float *h_A=(float*)malloc(size);
	float *h_B=(float*)malloc(size);
	float *h_C=(float*)malloc(size);
       	if (h_A == NULL || h_B == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // Initialize the host input vectors
        
        for (int i = 0; i < n*m; ++i)
        {
            scanf("%f",&h_A[i]);
            scanf("%f",&h_B[i]);
        }
       
       
        // Allocate the device input vector A
        float *d_A = NULL;

        // Allocate the device input vector B
        float *d_B = NULL;

        // Allocate the device output vector C
        float *d_C = NULL;
//	printf("mallocing done\n");

	hipMalloc((void **) &d_A,size);
	hipMalloc((void **) &d_B,size);
	hipMalloc((void **) &d_C,size);
        // Copy the host input vectors A and B in host memory to the device input vectors in
        // device memory

	hipMemcpy (d_A,h_A,size,hipMemcpyHostToDevice);
 	hipMemcpy (d_B,h_B,size,hipMemcpyHostToDevice);
	//cudaMemcpy (d_C,h_C,size,cudaMemcpyHostToDevice);

        // initialize blocksPerGrid and threads Per Block
	int blocksPerGrid=m;
	int threadsPerBlock=n;
//	printf("launchin the kernel\n");
        hadamard<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        hipMemcpy (h_C,d_C,size,hipMemcpyDeviceToHost);

        // Verify that the result vector is correct
        for (int i = 0; i < n*m; ++i)
        {
            if (fabs(h_A[i] * h_B[i] - h_C[i]) > 1e-5)
            {
               // fprintf(stderr, "Result verification failed at element %d!\n", i);
//                exit(EXIT_FAILURE);
            }
        }

        LOG printf("Test PASSED\n");

        // Free device global memory
        hipFree(h_A);
	hipFree(h_B);
	hipFree(h_C);

	free(h_A);
	free(h_B);
	//free(h_C);

        // Free host memory
        
        
        err = hipDeviceReset();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        print_matrix(h_C,m,n);
        
        LOG printf("Done\n");
    }
    return 0;
}

