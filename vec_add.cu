#include "hip/hip_runtime.h"
%%writefile vec_add.cu

#include <iostream>
#include <fstream>
#include <string>

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

__global__ void vector_add_kernel(const int* a, const int* b, int *c, int N){
    
    int thread_index = threadIdx.x;
    
    if (thread_index < N)
        c[thread_index] = a[thread_index] + b[thread_index];
}

void print_to_file(const char* file_name, const int* a, const int *b, const int *c, int N){
    std::ofstream fout(file_name);
    if (fout.is_open()){
        for (int i = 0; i < N; i++){
            fout << a[i] << " " << b[i] << " " << c[i] << "\n";
        }
        fout.close();
    }
    else {
        std::cout << "Unable to open file\n";
    }
}

int main(){
    hipError_t err = hipSuccess;
    int N = 512;
    size_t size = N * sizeof(int);

    int *a = new int[N];
    int *b = new int[N];
    int *c = new int[N];

    for (int i = 0; i < N; i++) {
        a[i] = rand() % 10 + 1;
        b[i] = rand() % 10 + 1;
        c[i] = 0;
    }

    print_to_file("input.txt", a, b, c, N);

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, size, hipMemcpyHostToDevice);
    
    vector_add_kernel<<<1, N>>> (dev_a, dev_b, dev_c, N);

    hipMemcpy(a, dev_a, size, hipMemcpyDeviceToHost);
    hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    print_to_file("output.txt", a, b, c, N);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    delete[] a;
    delete[] b;
    delete[] c;
    return 0;
}